#include "hip/hip_runtime.h"

#include "hipblas.h"

extern "C" void launch_cuBLAS_mat_mul(float *A, float *B, float *C, int m, int k, int n) {
    size_t bytes_A = m * k * sizeof(float);
    size_t bytes_B = k * n * sizeof(float);
    size_t bytes_C = m * n * sizeof(float);

    float *d_A;
    float *d_B;
    float *d_C;

    hipMalloc(&d_A, bytes_A);
    hipMalloc(&d_B, bytes_B);
    hipMalloc(&d_C, bytes_C);

    hipMemcpy(d_A, A, bytes_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bytes_B, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);

    hipMemcpy(C, d_C, bytes_C, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}